
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
typedef struct { float x, y, z, vx, vy, vz; } Body;
void randomizeBodies(float *data, int n) {
  for (int i = 0; i < n; i++) {
    data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
  }
}
__global__ void bodyForce(Body *p, float dt, int n,float softening) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;
    for (int j = 0; j < n; j++) {
      float dx = p[j].x - p[i].x;
      float dy = p[j].y - p[i].y;
      float dz = p[j].z - p[i].z;
      float distSqr = dx*dx + dy*dy + dz*dz + softening;
      float invDist = 1.0/sqrt(distSqr);
      float invDist3 = invDist * invDist * invDist;
      Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
    }
    p[i].vx += dt*Fx; p[i].vy += dt*Fy; p[i].vz += dt*Fz;
  }
}
__global__ void gpu_bodyForce(float *p, float dt, int n, float softening) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;
    for (int j = 0; j < n; j++) {
      float dx = p[6*j] - p[6*i];
      float dy = p[6*j+1] - p[6*i+1];
      float dz = p[6*j+2] - p[6*i+2];
      float distSqr = dx*dx + dy*dy + dz*dz + softening;
      float invDist = 1.0/sqrt(distSqr);
      float invDist3 = invDist * invDist * invDist;
      Fx += dx * invDist3; 
      Fy += dy * invDist3; 
      Fz += dz * invDist3;
    }
    p[6*i+3]+= dt*Fx; 
    p[6*i+4] += dt*Fy; 
    p[6*i+5] += dt*Fz;
  }
}
__global__ void gpu_integrate(float *p, float dt, int n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
      p[6*i] += p[6*i+3]*dt;
      p[6*i+1] += p[6*i+4]*dt;
      p[6*i+2] += p[6*i+5]*dt;
  }
}
int main(const int argc, const char** argv) {
  int user_value = atoi(argv[1]);
  int nBodies = user_value;
  int block_size =  128;
  float softening = 0.000000001;
  hipError_t nb_error;
  const float dt = 0.01; // time step
  int bytes = nBodies*sizeof(Body);
  float *h_buf = (float*)malloc(bytes);
  float *d_resp = (float*)malloc(bytes);
  randomizeBodies(h_buf, 6*nBodies); // Init pos / vel data
  float *d_buf;
  int nBlocks = (nBodies + block_size - 1) / block_size;
    float time;
    hipEvent_t start, stop;   
    hipEventCreate(&start) ;
    hipEventCreate(&stop) ;
    hipEventRecord(start, 0) ;
  hipMalloc(&d_buf, bytes);
  nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 1: %s\n", hipGetErrorString(nb_error));
  hipMemcpy(d_buf, h_buf, bytes, hipMemcpyHostToDevice);
   nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 2: %s\n", hipGetErrorString(nb_error));
    gpu_bodyForce<<<nBlocks, block_size>>>(d_buf, dt, nBodies,softening); // compute interbody forces
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 3: %s\n", hipGetErrorString(nb_error));
    hipDeviceSynchronize();
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 3: %s\n", hipGetErrorString(nb_error));
   gpu_integrate<<<nBlocks, block_size>>>(d_buf, dt, nBodies); // compute interbody forces
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 3: %s\n", hipGetErrorString(nb_error));
   hipMemcpy(d_resp, d_buf, bytes, hipMemcpyDeviceToHost);
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 4: %s\n", hipGetErrorString(nb_error));
    hipEventRecord(stop, 0) ;
    hipEventSynchronize(stop) ;
    hipEventElapsedTime(&time, start, stop) ;
    printf("cuda\t%d\t%3.1f\n", nBodies,time);
    free(h_buf);
    free(d_resp);
    hipFree(d_buf);
}
