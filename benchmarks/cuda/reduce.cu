
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define imin(a,b) (a<b?a:b)

const int threadsPerBlock = 256;



__global__ void dot(float* a, float* c, int N) {
	__shared__ float cache[threadsPerBlock];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int cacheIndex = threadIdx.x;

	float temp = 0;
	while (tid < N){
		temp += a[tid] ;
		tid += blockDim.x * gridDim.x;
	}

	cache[cacheIndex] = temp;

	__syncthreads();

	int i = blockDim.x/2;
	while (i != 0){
		if (cacheIndex < i)
			cache[cacheIndex] += cache[cacheIndex + i];

		__syncthreads();
		i /= 2;
	}

	if (cacheIndex == 0)
		c[blockIdx.x] = cache[0];
}

int main (int argc, char *argv[]) {
	float *a, c, *partial_c;
	float *dev_a, *dev_partial_c;
	
	int N = atoi(argv[1]);
   

	int blocksPerGrid = imin(32, (N+threadsPerBlock-1) / threadsPerBlock);

	a = (float*)malloc(N*sizeof(float));

	
	partial_c = (float*)malloc(blocksPerGrid*sizeof(float));

	for(int i=0; i<N; i++) {
		a[i] = i;
		
	}
    
	float time;
    hipEvent_t start, stop;   
    hipEventCreate(&start) ;
    hipEventCreate(&stop) ;
    hipEventRecord(start, 0) ;


	hipMalloc((void**)&dev_a, N*sizeof(float));
	
	hipMalloc((void**)&dev_partial_c, blocksPerGrid*sizeof(float));
	hipMemcpy(dev_a, a, N*sizeof(float), hipMemcpyHostToDevice);
	
	dot<<<blocksPerGrid, threadsPerBlock>>>(dev_a,  dev_partial_c, N);

	hipMemcpy(partial_c,dev_partial_c, blocksPerGrid*sizeof(float), hipMemcpyDeviceToHost);

	c = 0;
	for(int i=0; i<blocksPerGrid; i++) {
		c += partial_c[i];
	}
   
	hipFree(dev_a);
	hipFree(dev_partial_c);
    
	hipEventRecord(stop, 0) ;
    hipEventSynchronize(stop) ;
    hipEventElapsedTime(&time, start, stop) ;

    printf("CUDA\t%d\t%3.1f\n", N,time);

	//printf("\n FINAL RESULTADO: %f \n", c);

	free(a);
	free(partial_c);
  	
}