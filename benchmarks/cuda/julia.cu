
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <malloc.h>
#define _bitsperpixel 32
#define _planes 1
#define _compression 0

#define _xpixelpermeter 0x13B //0x130B //2835 , 72 DPI
#define _ypixelpermeter 0x13B//0x130B //2835 , 72 DPI
#define pixel 0xFF
#pragma pack(push,1)
typedef struct{
    uint8_t signature[2];
    uint32_t filesize;
    uint32_t reserved;
    uint32_t fileoffset_to_pixelarray;
} fileheader;
typedef struct{
    uint32_t dibheadersize;
    uint32_t width;
    uint32_t height;
    uint16_t planes;
    uint16_t bitsperpixel;
    uint32_t compression;
    uint32_t imagesize;
    uint32_t ypixelpermeter;
    uint32_t xpixelpermeter;
    uint32_t numcolorspallette;
    uint32_t mostimpcolor;
} bitmapinfoheader;
typedef struct {
    fileheader fileheader;
    bitmapinfoheader bitmapinfoheader;
} bitmap;
#pragma pack(pop)

__global__
void julia_kernel(float *ptr, int dim)
{
int x = blockIdx.x;
int y = blockIdx.y;
int offset = (x + (y * dim));
int juliaValue = 1;
float scale = 0.1;
float jx = ((scale * (dim - x)) / dim);
float jy = ((scale * (dim - y)) / dim);
float cr = (- 0.8);
float ci = 0.156;
float ar = jx;
float ai = jy;
for( int i = 0; i<200; i++){
float nar = (((ar * ar) - (ai * ai)) + cr);
float nai = (((ai * ar) + (ar * ai)) + ci);
if((((nar * nar) + (nai * nai)) > 1000))
{
	juliaValue = 0;
break;
}

	ar = nar;
	ai = nai;
}

	ptr[((offset * 4) + 0)] = (255 * juliaValue);
	ptr[((offset * 4) + 1)] = 0;
	ptr[((offset * 4) + 2)] = 0;
	ptr[((offset * 4) + 3)] = 255;
}



void genBpm (int height, int width, float *pixelbuffer_f) {
    uint32_t pixelbytesize = height*width*_bitsperpixel/8;
    uint32_t  _filesize =pixelbytesize+sizeof(bitmap);
    FILE *fp = fopen("test.bmp","wb");
    bitmap *pbitmap  = (bitmap*)calloc(1,sizeof(bitmap));

    int buffer_size = height*width*4;
    uint8_t *pixelbuffer = (uint8_t*)malloc(buffer_size);

    for(int i = 0; i<buffer_size;i++)
    {
     pixelbuffer[i]= (uint8_t) pixelbuffer_f[i];
    }


    //strcpy(pbitmap->fileheader.signature,"BM");
    pbitmap->fileheader.signature[0] = 'B';
    pbitmap->fileheader.signature[1] = 'M';
    pbitmap->fileheader.filesize = _filesize;
    pbitmap->fileheader.fileoffset_to_pixelarray = sizeof(bitmap);
    pbitmap->bitmapinfoheader.dibheadersize =sizeof(bitmapinfoheader);
    pbitmap->bitmapinfoheader.width = width;
    pbitmap->bitmapinfoheader.height = height;
    pbitmap->bitmapinfoheader.planes = _planes;
    pbitmap->bitmapinfoheader.bitsperpixel = _bitsperpixel;
    pbitmap->bitmapinfoheader.compression = _compression;
    pbitmap->bitmapinfoheader.imagesize = pixelbytesize;
    pbitmap->bitmapinfoheader.ypixelpermeter = _ypixelpermeter ;
    pbitmap->bitmapinfoheader.xpixelpermeter = _xpixelpermeter ;
    pbitmap->bitmapinfoheader.numcolorspallette = 0;
    fwrite (pbitmap, 1, sizeof(bitmap),fp);
    //memset(pixelbuffer,pixel,pixelbytesize);
    fwrite(pixelbuffer,1,pixelbytesize,fp);
    fclose(fp);
    free(pbitmap);
    free(pixelbuffer);
}


int main( void ) {
   
    int height = 1000;
    int width  = 1000;
    int DIM = 1000;
    int size_array = height*width*4*sizeof(float);
    hipError_t j_error;
    
    //int pixelbytesize=  height*width*_bitsperpixel/8;
    //printf(" pixel byte size %lu\n",pixelbytesize);
   
     float *h_pixelbuffer = (float*)malloc(size_array);
     float *d_pixelbuffer;

     ////////
    hipMalloc( (void**)&d_pixelbuffer, size_array);
    j_error = hipGetLastError();
    if(j_error != hipSuccess) printf("Error 1: %s\n", hipGetErrorString(j_error));
    ////////

    
    ////////////////////
    dim3 grid(DIM,DIM);

    julia_kernel<<<grid, 1>>>(d_pixelbuffer,height); // compute interbody forces
    j_error = hipGetLastError();
    if(j_error != hipSuccess) printf("Error 3: %s\n", hipGetErrorString(j_error));
  ////////



    hipMemcpy(h_pixelbuffer, d_pixelbuffer, size_array, hipMemcpyDeviceToHost); // return results 
    j_error = hipGetLastError();
    if(j_error != hipSuccess) printf("Error 7: %s\n", hipGetErrorString(j_error));


    //for(int i=0;i<pixelbytesize; i++)
      //     printf("pixel %d = %d\n",i,pixelbuffer[i]);
    
    genBpm(height,width,h_pixelbuffer);
   
    free(h_pixelbuffer);
    hipFree(d_pixelbuffer);
}



